#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdint.h>
#include <stdio.h>

#include "nvbit_reg_rw.h"
#include "utils/utils.h"
#include "pf_injector.h"
#include "arch.h"


extern "C" __device__ __noinline__ void inject_error(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int maxRegs) {

				inj_info_t* inj_info = (inj_info_t*)piinfo; 
				uint32_t verbose_device = *((uint32_t *)pverbose_device);

				uint32_t smid;
				asm("mov.u32 %0, %smid;" :"=r"(smid));
				if (smid != inj_info->injSMID) 
								return; // This is not the selected SM. No need to proceed.

				uint32_t laneid;
				asm("mov.u32 %0, %laneid;" :"=r"(laneid));
				if (laneid != inj_info->injLaneID) 
								return; // This is not the selected Lane ID. No need to proceed.

				assert(numDestGPRs > 0);
				uint32_t injAfterVal = 0; 
				uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
				if (DUMMY) {
								injAfterVal = injBeforeVal;
				} else {
								injAfterVal = injBeforeVal ^ inj_info->injMask; 
								nvbit_write_reg(destGPRNum, injAfterVal);
				}
				// updating counter/flag to check whether the error was injected
				if (verbose_device) printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x\n", destGPRNum, injBeforeVal, nvbit_read_reg(destGPRNum), injAfterVal);
				inj_info->errorInjected = true; 
				atomicAdd((unsigned long long*) &inj_info->injNumActivations, 1LL);  
}



extern "C" __device__ __noinline__ void clock_block(clock_t *d_o, clock_t clock_count)
{
    clock_t start_clock = clock();
    clock_t clock_offset = 0;
    while (clock_offset < clock_count)
    {
        clock_offset = clock() - start_clock;
    }
     d_o[0] = clock_offset;
	 clock_t now = clock();
	//  printf("Clock start_clock %lu clock_offset %lu now %lu clock_count %lu\n", start_clock, clock_offset, now, clock_count);
}

extern "C" __device__ __noinline__ void print_op(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int maxRegs) {

				// printf("blockDim.x %d gridDim.x %d threadIdx.x %d blockIdx.x %d\n", blockDim.x, gridDim.x, threadIdx.x, blockIdx.x);
				inj_info_t* inj_info = (inj_info_t*)piinfo; 
				uint32_t verbose_device = *((uint32_t *)pverbose_device);
				clock_t cur = clock();
							
				assert(numDestGPRs > 0);
				clock_block(&cur, 1 0);
				//printf("Clock clock %lu \n", cur);
				inj_info->clock = cur;
				// uint32_t smid;
				// asm("mov.u32 %0, %smid;" :"=r"(smid));
				// if (smid != inj_info->injSMID) 
				// 				return; // This is not the selected SM. No need to proceed.

				// uint32_t laneid;
				// asm("mov.u32 %0, %laneid;" :"=r"(laneid));
				// if (laneid != inj_info->injLaneID) 
				//              return; // This is not the selected Lane ID. No need to proceed.

				// printf("Clock clock %lu \n", cur);
				// uint32_t injAfterVal = 0; 
				// uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
				// if (DUMMY) {
				// 				injAfterVal = injBeforeVal;
				// } else {
				// 				injAfterVal = injBeforeVal ^ inj_info->injMask; 
				// 				nvbit_write_reg(destGPRNum, injAfterVal);
				// }
				// updating counter/flag to check whether the error was injected
				// if (verbose_device) printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x instType=%lu\n", destGPRNum, injBeforeVal, nvbit_read_reg(destGPRNum), injAfterVal, inj_info->injInstType);
				// inj_info->errorInjected = true; 
				// atomicAdd((unsigned long long*) &inj_info->injNumActivations, 1LL);  
}

