#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdint.h>
#include <stdlib.h>
#include <math.h>

static const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}


#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void access_memory(ulong* memory, ulong* offset, int count){
    int global_id = threadIdx.x + blockDim.x * blockIdx.x;
    ulong *p = memory;
    p += offset[global_id];

    ulong q;
    for(int i=0; i<count; i++){
        q = (ulong)memory + *p;
        p = (ulong*)q; 
    }
    *p = 1;
}

int draw(int limit) { return rand() % limit; }

void swap(uint64_t *a, uint64_t *b) {
    uint64_t temp = *a;
    *a = *b;
    *b = temp;
}

uint64_t getTimeInNSecs() {
    struct timespec time;
    clock_gettime(CLOCK_MONOTONIC, &time);
    uint64_t timeInSec = time.tv_sec * 1e9 + time.tv_nsec;
    return timeInSec;
}

int main(int argc, char **argv){
    uint64_t start, end, kernel_time;
    int len, size;
    unsigned int count;
    size_t grid_size, workgroup_size;

    if (argc != 5) {
        printf("./kernel memory_size num_wg count\n");
        exit(-1);
    }

    size = atoi(argv[1]);
    count = atoi(argv[3]);
    int num_wg = atoi(argv[2]);
    int gpuID = atoi(argv[4]);
    int num_wi_per_wg = 32;

    grid_size = num_wg * num_wi_per_wg;
    workgroup_size = num_wi_per_wg;

    len = size / sizeof(void *);
    int M = pow(2, 24);

    /*P2P*/
    hipDeviceProp_t prop[64];
    checkCudaErrors(hipGetDeviceProperties(&prop[0], 0));
    checkCudaErrors(hipGetDeviceProperties(&prop[1], 1));
    int p2pCapableGPUs[2]; 
    p2pCapableGPUs[0] = 0;
    p2pCapableGPUs[1] = 1;
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    checkCudaErrors(hipDeviceEnablePeerAccess(p2pCapableGPUs[1], 0));
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    checkCudaErrors(hipDeviceEnablePeerAccess(p2pCapableGPUs[0], 0));


    /* Host*/
    uint64_t *memory = new uint64_t[M * sizeof(uint64_t)]; //malloc(M * sizeof(uint64_t));
    uint64_t *indices = new uint64_t[M * sizeof(uint64_t)]; //malloc(M * sizeof(uint64_t));
    for (int i = 0; i < len; i++) {
        indices[i] = i;
    }
    for (int i = 0; i < len - 1; i++) {
        int j = i + draw(len - i);
        if (j != i) {
            swap(&indices[i], &indices[j]);
        }
    }

    for (int i = 1; i < len; i++) {
        memory[indices[i - 1]] = indices[i] * 8;
    }
    memory[indices[len - 1]] = indices[0] * 8;

    /* Data copy */
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    uint64_t *d_memory;
    checkCudaErrors(hipMalloc ((void **) &d_memory, sizeof(uint64_t) * M));
    checkCudaErrors(hipMemcpy((void *)d_memory, memory, sizeof(uint64_t) * M, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    uint64_t *d_offset;
    checkCudaErrors(hipMalloc ((void **) &d_offset, sizeof(uint64_t) * grid_size));
    uint64_t zero = 0;
    checkCudaErrors(hipMemset((void *)d_offset, zero, sizeof(uint64_t) * grid_size));


    /* Execute */
    start = getTimeInNSecs();
    access_memory<<<num_wg, num_wi_per_wg>>>(d_memory, d_offset, count);
    hipDeviceSynchronize ();

    hipError_t error_id = hipGetLastError();
    if (error_id != hipSuccess)
    {
        printf("Error is %s\n", hipGetErrorString(error_id));
    }

    end = getTimeInNSecs();
    kernel_time = end - start;
    printf("GPU Runtime: %.10f %i\n", (double)kernel_time / count, size);
    
    hipFree(d_memory);
    hipFree(d_offset);
    delete[] memory;
    delete[] indices;
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    checkCudaErrors(hipDeviceDisablePeerAccess(p2pCapableGPUs[1]));
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    checkCudaErrors(hipDeviceDisablePeerAccess(p2pCapableGPUs[0]));

    return 0;
}