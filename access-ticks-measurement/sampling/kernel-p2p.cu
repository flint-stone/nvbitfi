#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdint.h>
#include <stdlib.h>
#include <math.h>

static const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}


#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void access_memory(ulong* memory, ulong* offset, int count){
    int global_id = threadIdx.x + blockDim.x * blockIdx.x;
    ulong *p = memory;
    p += offset[global_id];

    ulong q;
    for(int i=0; i<count; i++){
        q = (ulong)memory + *p;
        p = (ulong*)q; 
    }
    if(global_id == -1) *p = 1;
}

__global__ void access_memory_std(ulong* memory, ulong* offset, int count, unsigned long long int* d_time, double* avg_arr, double* std_arr){
    unsigned long long int start_time, end_time;
    int global_id = threadIdx.x + blockDim.x * blockIdx.x;
    
    ulong *p = memory;
    p += offset[global_id];
    int global_id_time = global_id * count;
    ulong q;
    ulong sum = 0;
    for(int i=0; i<count; i++){
        start_time = clock(); //clock64();
        q = (ulong)memory + *p;
        p = (ulong*)q; 
        end_time = clock(); //clock64();
        //printf("global_id 1 %d d_time %lli\n", global_id, d_time);
        d_time[global_id_time++] = end_time - start_time;
        //printf("global_id 2 %d %llu\n", global_id_time, end_time - start_time);
        sum += end_time - start_time;
    }
    
    double avg = sum /(double)count;
    double sqr_sum = 0.0;
    global_id_time = global_id * count;
    for(int i = 0; i < count ; i++){
        double diff = d_time[global_id_time + i] - avg;
        //printf("diff %f\n", diff);
        sqr_sum += diff * diff;
    }
    double std = sqrt((double)sqr_sum /count);
    avg_arr[global_id] = avg;
    std_arr[global_id] = std;
    *p = 1;
}

int draw(int limit) { return rand() % limit; }

void swap(uint64_t *a, uint64_t *b) {
    uint64_t temp = *a;
    *a = *b;
    *b = temp;
}

uint64_t getTimeInNSecs() {
    struct timespec time;
    clock_gettime(CLOCK_MONOTONIC, &time);
    uint64_t timeInSec = time.tv_sec * 1e9 + time.tv_nsec;
    return timeInSec;
}

int main(int argc, char **argv){
    uint64_t start, end, kernel_time;
    int len, size;
    unsigned int count;
    size_t grid_size, workgroup_size;
    bool use_std = false;

    // if (argc != 6) {
    //     printf("./kernel-p2p memory_size num_wg count\n");
    //     exit(-1);
    // }

    size = atoi(argv[1]);
    count = atoi(argv[3]);
    int num_wg = atoi(argv[2]);
    int gpuID = atoi(argv[4]);
    int peerGpuID = atoi(argv[5]);
    int num_wi_per_wg = atoi(argv[5]);

    grid_size = num_wg * num_wi_per_wg;
    workgroup_size = num_wi_per_wg;

    len = size / sizeof(void *);
    int M = pow(2, 32);

    /*P2P*/
    hipDeviceProp_t prop[64];
    checkCudaErrors(hipGetDeviceProperties(&prop[0], gpuID));
    checkCudaErrors(hipGetDeviceProperties(&prop[1], peerGpuID));
    int p2pCapableGPUs[2]; 
    p2pCapableGPUs[0] = gpuID;
    p2pCapableGPUs[1] = peerGpuID;
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    checkCudaErrors(hipDeviceEnablePeerAccess(p2pCapableGPUs[1], 0));
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    checkCudaErrors(hipDeviceEnablePeerAccess(p2pCapableGPUs[0], 0));


    /* Host*/
    uint64_t *memory = new uint64_t[M * sizeof(uint64_t)]; //malloc(M * sizeof(uint64_t));
    uint64_t *indices = new uint64_t[M * sizeof(uint64_t)]; //malloc(M * sizeof(uint64_t));
    uint64_t *offset = new uint64_t[grid_size * sizeof(uint64_t)]; //malloc(M * sizeof(uint64_t));
    for (int i = 0; i < len; i++) {
        indices[i] = i;
    }
    for (int i = 0; i < len - 1; i++) {
        int j = i + draw(len - i);
        if (j != i) {
            swap(&indices[i], &indices[j]);
        }
    }

    for (int i = 1; i < len; i++) {
        memory[indices[i - 1]] = indices[i] * 8;
    }
    memory[indices[len - 1]] = indices[0] * 8;

    for(int i = 0; i < grid_size; i++){
        offset[i] = i;
    }

    /* Data copy */
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    uint64_t *d_memory;
    checkCudaErrors(hipMalloc ((void **) &d_memory, sizeof(uint64_t) * M));
    checkCudaErrors(hipMemcpy((void *)d_memory, memory, sizeof(uint64_t) * M, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    uint64_t *d_offset;
    checkCudaErrors(hipMalloc ((void **) &d_offset, sizeof(uint64_t) * grid_size));
    uint64_t zero = 0;
    // checkCudaErrors(hipMemset((void *)d_offset, zero, sizeof(uint64_t) * grid_size));
    checkCudaErrors(hipMemcpy((void *)d_offset, offset, sizeof(uint64_t) * grid_size, hipMemcpyHostToDevice));


    /* Execute */
    if(use_std){
        unsigned long long int *d_time;
        double  *d_avg_arr;
        double *d_std_arr;
        hipMalloc ((void **) &d_time, sizeof(unsigned long long int) * count * num_wi_per_wg);
        hipMemset((void *)d_time, (unsigned long long int)1, sizeof(unsigned long long int) * count * num_wi_per_wg);
        hipMalloc ((void **) &d_avg_arr, sizeof(double) * num_wi_per_wg);
        hipMemset((void *)d_avg_arr, 0.0, sizeof(double) * num_wi_per_wg);
        hipMalloc ((void **) &d_std_arr, sizeof(double) * num_wi_per_wg);
        hipMemset((void *)d_std_arr, 0.0, sizeof(double) * num_wi_per_wg);

        start = getTimeInNSecs();
        access_memory_std<<<num_wg, num_wi_per_wg>>>(d_memory, d_offset, count, d_time, d_avg_arr, d_std_arr);
        hipDeviceSynchronize ();

        hipError_t error_id = hipGetLastError();
        if (error_id != hipSuccess)
        {
            printf("Error is %s\n", hipGetErrorString(error_id));
        }

        end = getTimeInNSecs();
        double *h_avg_arr = new double[num_wi_per_wg]; 
        double *h_std_arr = new double[num_wi_per_wg]; 
        hipMemcpy((void *)h_avg_arr, d_avg_arr, sizeof(double) * num_wi_per_wg, hipMemcpyDeviceToHost);
        hipMemcpy((void *)h_std_arr, d_std_arr, sizeof(double) * num_wi_per_wg, hipMemcpyDeviceToHost);
        double sum_mean = 0.0;
        double sum_std = 0.0;
        for(int i = 0; i < num_wi_per_wg; i++){
            sum_mean += h_avg_arr[i];
            sum_std += h_std_arr[i];
        }
        printf("GPU Runtime: %.10f %.10f %i\n", (double)sum_mean / num_wi_per_wg, (double) sum_std / num_wi_per_wg, size);
        hipFree(d_time);
        hipFree(d_avg_arr);
        hipFree(d_std_arr);
        delete[] h_avg_arr;
        delete[] h_std_arr;
    }
    else{
        start = getTimeInNSecs();
        access_memory<<<num_wg, num_wi_per_wg>>>(d_memory, d_offset, count);
        hipDeviceSynchronize ();

        hipError_t error_id = hipGetLastError();
        if (error_id != hipSuccess)
        {
            printf("Error is %s\n", hipGetErrorString(error_id));
        }

        end = getTimeInNSecs();
        kernel_time = end - start;
        printf("GPU Runtime: %.10f %i\n", (double)kernel_time / count, size);
    }
    
    hipFree(d_memory);
    hipFree(d_offset);
    delete[] memory;
    delete[] indices;
    delete[] offset;
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[0]));
    checkCudaErrors(hipDeviceDisablePeerAccess(p2pCapableGPUs[1]));
    checkCudaErrors(hipSetDevice(p2pCapableGPUs[1]));
    checkCudaErrors(hipDeviceDisablePeerAccess(p2pCapableGPUs[0]));

    return 0;
}