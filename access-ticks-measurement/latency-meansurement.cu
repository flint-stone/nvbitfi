#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdint.h>

#include "repeat.h"

typedef unsigned long long int ptrsize_type;

__global__ void global_latency (ptrsize_type** my_array, int array_length, int iterations, unsigned long long * duration) {

    unsigned long long int start_time, end_time;
    ptrsize_type *j = (ptrsize_type*)my_array;
    volatile unsigned long long int sum_time;

    sum_time = 0;

    for (int k = 0; k < iterations; k++)
    {

        start_time = clock64();
        repeat1024(j=*(ptrsize_type **)j;)
        end_time = clock64();

        sum_time += (end_time - start_time);
    }

    ((ptrsize_type*)my_array)[array_length] = (ptrsize_type)j;
    ((ptrsize_type*)my_array)[array_length+1] = (ptrsize_type) sum_time;
    duration[0] = sum_time;
}

void parametric_measure_global(int N, int iterations, unsigned long long int maxMem, int stride)
{
    unsigned long long int maxMemToArraySize = maxMem / sizeof( ptrsize_type );
    unsigned long long int maxArraySizeNeeded = 1024*iterations*stride;
    unsigned long long int maxArraySize = (maxMemToArraySize<maxArraySizeNeeded)?(maxMemToArraySize):(maxArraySizeNeeded);

    ptrsize_type* h_a = new ptrsize_type[maxArraySize+2];
    ptrsize_type** d_a;
    hipMalloc ((void **) &d_a, (maxArraySize+2)*sizeof(ptrsize_type));

    unsigned long long int* duration;
    hipMalloc ((void **) &duration, sizeof(unsigned long long int));

    for ( int i = 0; true; i += stride)
    {
        ptrsize_type nextAddr = ((ptrsize_type)d_a)+(i+stride)*sizeof(ptrsize_type);
        if( i+stride < maxArraySize )
        {
            printf("Initialize entry %i, next addr %lli\n", i, nextAddr);
            h_a[i] = nextAddr;
        }
        else
        {
            h_a[i] = (ptrsize_type)d_a; // point back to the first entry
            break;
        }
    }
    hipMemcpy((void *)d_a, h_a, (maxArraySize+2)*sizeof(ptrsize_type), hipMemcpyHostToDevice);

    unsigned long long int latency_sum = 0;
    int repeat = 1;
    for (int l=0; l <repeat; l++)
    {
        global_latency<<<1,1>>>(d_a, maxArraySize, iterations, duration);
        hipDeviceSynchronize ();

        hipError_t error_id = hipGetLastError();
        if (error_id != hipSuccess)
        {
            printf("Error is %s\n", hipGetErrorString(error_id));
        }

        unsigned long long int latency;
        hipMemcpy( &latency, duration, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
        latency_sum += latency;
    }

    hipFree(d_a);
    hipFree(duration);

    delete[] h_a;
    printf("latency average %f\n", (double)(latency_sum/(repeat*1024.0*iterations)) );
}

void measure_global_latency()
{
    int maxMem = 1024*1024*1024; // 1GB
    int N = 1024;
    int iterations = 1;

    for (int stride = 1; stride <= 1024; stride+=1)
    {
        printf (" stride_size  %5d, ", stride*sizeof( ptrsize_type ));
        parametric_measure_global( N, iterations, maxMem, stride );
    }
    for (int stride = 1024; stride <= 1024*1024; stride+=1024)
    {
        printf (" big_stride_size %5d, ", stride*sizeof( ptrsize_type ));
        parametric_measure_global( N, iterations, maxMem, stride );
    }
}

int main()
{
    measure_global_latency();
    return 0;
}